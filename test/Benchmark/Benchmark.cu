#include "hip/hip_runtime.h"
#define NR_BITS 16
#define NR_CHANNELS 480
#define NR_POLARIZATIONS 2
#define NR_SAMPLES_PER_CHANNEL 3072
#define NR_RECEIVERS 576
#define NR_BASELINES ((NR_RECEIVERS) * ((NR_RECEIVERS) + 1) / 2)
#define NR_RECEIVERS_PER_BLOCK 64
#define NR_TIMES_PER_BLOCK (128 / (NR_BITS))

#include "libtcc/Correlator.h"


#include <iostream>
#include <complex>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

inline void checkCudaCall(hipError_t error) 
{
	if (error != hipSuccess) { 
		std::cerr << "error " << error << std::endl;
		exit(1);
	}
}

typedef std::complex<__half>	Sample;
typedef std::complex<float>		Visibility;

typedef Sample Samples[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_RECEIVERS][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK];
typedef Visibility Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];

int main(int argc, char* argv[]) {	
	std::cout << "Benchmarking in progress..." << std::endl;
	
	try {
		checkCudaCall(hipSetDevice(0)); // combine the CUDA rutime API and CUDA driver API
		checkCudaCall(hipFree(0));
		
		// create correlator 
		tcc::Correlator correlator(NR_BITS, NR_RECEIVERS, NR_CHANNELS, NR_SAMPLES_PER_CHANNEL, NR_POLARIZATIONS, NR_RECEIVERS_PER_BLOCK);
		
		hipStream_t stream;
		Samples *samples;
		Visibilities *visibilities;
		
		// create stream and allocate memory 
		checkCudaCall(hipStreamCreate(&stream));
		checkCudaCall(hipMallocManaged(&samples, sizeof(Samples)));
		checkCudaCall(hipMallocManaged(&visibilities, sizeof(Visibilities)));
		
		// initialise 2 values at the input as (2+3i) and (4+5i)
		(*samples)[NR_CHANNELS / 3][NR_SAMPLES_PER_CHANNEL / 5 / NR_TIMES_PER_BLOCK][174][0][NR_SAMPLES_PER_CHANNEL / 5 % NR_TIMES_PER_BLOCK] = Sample(2, 3);
		(*samples)[NR_CHANNELS / 3][NR_SAMPLES_PER_CHANNEL / 5 / NR_TIMES_PER_BLOCK][418][0][NR_SAMPLES_PER_CHANNEL / 5 % NR_TIMES_PER_BLOCK] = Sample(4, 5);
	
		// run correlation operation
		correlator.launchAsync((hipStream_t) stream, (hipDeviceptr_t) visibilities, (hipDeviceptr_t) samples);
		checkCudaCall(hipDeviceSynchronize());
		
		// at this particular output cell (2+3i)(4-5i) = (23-2i)
		// conjugated as we are correlating
		std::cout << ((*visibilities)[160][87745][0][0] == Visibility(23, 2) ? "success" : "failed") << std:: endl;
		
		checkCudaCall(hipFree(visibilities));
		checkCudaCall(hipFree(samples));
		checkCudaCall(hipStreamDestroy(stream));
	} catch (std::exception &error) {
		std::cerr << error.what() << std::endl;
	}
	return 0;
}
